#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>

__global__ void sigmoidActivation(float *z_matrix, float *activation_matrix) {
    int index = threadIdx.x;
    activation_matrix[index] = 1.0 / (1.0 + exp(-z_matrix[index]));
}


int main()
{
    const int arraySize = 5;

    // Initialize arrays on the cpu 
    float host_z_values[arraySize] = { 1.0, 2.0, 3.0, 4.0, 5.0 };
    float host_activations[arraySize] = { 0.0 };

    // Caculate the number of bytes required to store the arrays on gpu
    const size_t bytes_z_values = arraySize * sizeof(float);
    const size_t bytes_activations = arraySize * sizeof(float);

    // Declare pointers for gpu based params
    float *device_z_values;
    float *device_activations;

    // Allocate memory on the gpu
    hipMalloc(&device_z_values, bytes_z_values);
    hipMalloc(&device_activations, bytes_activations);

    // Copy the arrays from cpu to gpu
    hipMemcpy(device_z_values, host_z_values, bytes_z_values, hipMemcpyHostToDevice);

    //Call the kernel, with 1 block and arraySize threads
    sigmoidActivation << <1, arraySize >> > (device_z_values, device_activations);

    // Copy the results from gpu to cpu
    hipMemcpy(host_activations, device_activations, bytes_activations, hipMemcpyDeviceToHost);

    printf("sigmoid({1,2,3,4,5}) = {%f, %f, %f, %f, %f}\n",
		host_activations[0],
		host_activations[1],
		host_activations[2],
		host_activations[3],
        host_activations[4]);
    //getchar();

    // Free the memory on gpu
    hipFree(device_z_values);
    hipFree(device_activations);


    return 0;
}
